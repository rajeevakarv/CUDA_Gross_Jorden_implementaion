#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include "common.h"

const int TILE_SIZE = 256;          //Max no of threads in Block.
const int MAX_GRID_SIZE = 65535;    //Max no of Blocks  in a Grid.

// GPU Kernel-1 to perform row scaling.
__global__ void GaussianEliminationGPUKernelScaling(float* matrix, unsigned int numberOfRows, unsigned int numberOfColumns, float* outputMatrix, bool partialPivot, unsigned int row)
{
	// Retrieve our coordinates in the block
	int blockId   = blockIdx.y * gridDim.x + blockIdx.x;			 	
	int threadId = blockId * blockDim.x + threadIdx.x;   //Thread id calculation

	if(threadId<numberOfColumns*numberOfRows){
		if ((threadId/numberOfColumns)==row)
			outputMatrix[threadId] = matrix[threadId] / matrix[numberOfColumns*row+row];
		else 
			outputMatrix[threadId] = matrix[threadId];
	}
}

// GPU Kernel-2 to perform the reduction in rows.
__global__ void GaussianEliminationGPUKernelReduction(float* Matrix, unsigned int numberOfRows, unsigned int numberOfColumns, float* outputMatrix, bool partialPivot, unsigned int row)
{
	// Retrieve our coordinates in the block
	int blockId   = blockIdx.y * gridDim.x + blockIdx.x;			 	
	int threadId = blockId * blockDim.x + threadIdx.x;     //threadid calculation

	float pivot = Matrix[numberOfColumns*row+row];         //Calculates the pivot element for each row.
	if(threadId<numberOfColumns*numberOfRows){
		if ((threadId/numberOfColumns)!=row)
			outputMatrix[threadId] = Matrix[threadId]- Matrix[(threadId/numberOfColumns)*numberOfRows+row] * (Matrix[row*numberOfColumns + threadId%numberOfColumns]/pivot);
		else 
			outputMatrix[threadId] = Matrix[threadId];
		}
}

// GPU function for direct method Gross Jorden method.

bool GaussianEliminationGPU( float** matrix, unsigned int numberOfRows, unsigned int numberOfColumns, float** outputMatrix, bool partialPivot)
{
	// Error return value
	hipError_t status;
	// Number of bytes in the matrix.
	int bytes = numberOfColumns * numberOfRows *sizeof(float);

	unsigned int numberOfRowsd, numberOfColumnsd;    //To be safe copy the elements too.
	numberOfColumnsd = numberOfColumns;
	numberOfRowsd = numberOfRows;

	// Pointers to the device arrays
	float *matrixd, *outputMatrixd;                    //input and output matrix
	// Allocate memory on the device to store each matrix
	hipMalloc((void**) &matrixd, bytes);
	status = hipGetLastError();              //To check the error
	if (status != hipSuccess) {                     
		std::cout << "Kernel failed2: " << hipGetErrorString(status) << 
		std::endl;
		hipFree(matrixd);                     //Free call for memory
		hipFree(outputMatrixd);               //Free call for memory
		return false;
	}

	hipMalloc((void**) &outputMatrixd, bytes);
	status = hipGetLastError();              //To check the error
	if (status != hipSuccess) {                     
		std::cout << "Kernel failed2: " << hipGetErrorString(status) << 
		std::endl;
		hipFree(matrixd);                     //Free call for memory
		hipFree(outputMatrixd);               //Free call for memory
		return false;
	}
	
	float *temp1 = matrixd;
	float *temp2 = outputMatrixd;

	// Copy the host input data to the device
	for (int i=0; i<numberOfRows; i++){
		hipMemcpy((float *)temp1, matrix[i], numberOfColumns *sizeof(float), hipMemcpyHostToDevice);
		status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Kernel failed3: " << hipGetErrorString(status) << 
			std::endl;
			hipFree(matrixd);                   //Free call for memory
			hipFree(outputMatrixd);              //Free call for memory
			return false;
		}
		hipMemcpy((float *)temp2, matrix[i], numberOfColumns *sizeof(float), hipMemcpyHostToDevice);
		status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Kernel failed4: " << hipGetErrorString(status) << 
			std::endl;
			hipFree(matrixd);                      //Free call for memory
			hipFree(outputMatrixd);                //Free call for memory
			return false; 
		}
		temp1 += numberOfColumns;
		temp2 += numberOfColumns;
	}
	temp1= matrixd;
	temp2 = outputMatrixd;
	
	int size = numberOfColumns * numberOfRows;
	dim3 dimBlock(TILE_SIZE, 1);
	int gridx = 1;                      //Grid size calculation
	int gridy = 1;                      //Grid size calculation
	if(size/TILE_SIZE < MAX_GRID_SIZE)
		gridx = ceil((float)size/TILE_SIZE);            //Decide the grid size for input size.
	else{
		gridx = MAX_GRID_SIZE;
		gridy = ceil((float)size/(TILE_SIZE*MAX_GRID_SIZE));
	}

	dim3 dimGrid(gridx, gridy); // grid call.
	
	// Launch the kernel one-by-one
	int rowNo = 0; 
	for (rowNo=0; rowNo < numberOfColumns ;rowNo++){
		GaussianEliminationGPUKernelScaling<<<dimGrid, dimBlock>>>(matrixd, numberOfRowsd, numberOfColumnsd, outputMatrixd, partialPivot, rowNo);    //Calling kernel-1 for scaling
		hipDeviceSynchronize();                //Thread sync
		status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Kernel failed5: " << hipGetErrorString(status) << 
			std::endl;
			hipFree(matrixd);                   //Free call for memory
			hipFree(outputMatrixd);             //Free call for memory
			return false;
		}
		
		GaussianEliminationGPUKernelReduction<<<dimGrid, dimBlock>>>(outputMatrixd, numberOfRowsd, numberOfColumnsd, matrixd, partialPivot, rowNo);       //Calling kernel-2 for reduction
		status = hipGetLastError();     //Error check
		if (status != hipSuccess) {
			std::cout << "Kernel failed6: " << hipGetErrorString(status) << 
			std::endl;
			hipFree(matrixd);                    //Free call for memory
			hipFree(outputMatrixd);              //Free call for memory
			return false;
		}

		hipDeviceSynchronize();          //thread sync
	}

	// Check for errors
	status = hipGetLastError();
	if (status != hipSuccess) {
		std::cout << "Kernel failed7: " << hipGetErrorString(status) << 
		std::endl;
		hipFree(matrixd);                      //Free call for memory
		hipFree(outputMatrixd);                //Free call for memory
		return false;
	}
	// Retrieve the result matrix
	for (int i=0; i<numberOfRows; i++){
		hipMemcpy(outputMatrix[i], matrixd, numberOfColumns *sizeof(float), hipMemcpyDeviceToHost);
		matrixd += numberOfColumns;
	}
	// Free device memory
	hipFree(outputMatrixd);                       //Free call for memory
	hipFree(matrixd);                             //Free call for memory
	// Success
	return true;
}
